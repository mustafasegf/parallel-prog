#include "hip/hip_runtime.h"
#include "matrix.hpp"
#include <chrono>
#include <hipblas.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <immintrin.h>
#include <iomanip>
#include <iostream>
#include <numeric>
#include <stdint.h>
#include <stdio.h>

using data_type = double;
// constexpr int32_t TILE_WIDTH = 1024 / sizeof(data_type);
constexpr int32_t TILE_WIDTH = 32;

__global__ void matrixMulKernel(const data_type *matrix1,
                                const data_type *matrix2, data_type *answer,
                                int32_t rows1, int32_t cols1, int32_t cols2) {

  int32_t row = blockIdx.y * blockDim.y + threadIdx.y;
  int32_t col = blockIdx.x * blockDim.x + threadIdx.x;
  data_type sum = 0;

  if (row < rows1 && col < cols2) {
    for (int32_t i = 0; i < cols1; i++) {
      sum += matrix1[row * cols1 + i] * matrix2[i * cols2 + col];
    }
    answer[row * cols2 + col] = sum;
  }
}

__global__ void matrixMulSharedKernel(const data_type *matrix1,
                                      const data_type *matrix2,
                                      data_type *answer, int32_t rows1,
                                      int32_t cols1, int32_t cols2) {

  __shared__ data_type shared_matrix1[TILE_WIDTH][TILE_WIDTH];
  __shared__ data_type shared_matrix2[TILE_WIDTH][TILE_WIDTH];

  int32_t row = blockIdx.y * blockDim.y + threadIdx.y;
  int32_t col = blockIdx.x * blockDim.x + threadIdx.x;

  data_type sum = 0;

  shared_matrix1[threadIdx.y][threadIdx.x] = 0;
  shared_matrix2[threadIdx.y][threadIdx.x] = 0;

  if (row < rows1 && col < cols2) {
    for (int32_t i = 0; i < cols1; i += TILE_WIDTH) {
      shared_matrix1[threadIdx.y][threadIdx.x] =
          matrix1[row * cols1 + i + threadIdx.x];
      shared_matrix2[threadIdx.y][threadIdx.x] =
          matrix2[(i + threadIdx.y) * cols2 + col];
      __syncthreads();

      for (int32_t j = 0; j < TILE_WIDTH; j++) {
        sum += shared_matrix1[threadIdx.y][j] * shared_matrix2[j][threadIdx.x];
      }
      __syncthreads();
    }
    answer[row * cols2 + col] = sum;
  }
}

int main(int argc, char *argv[]) {
  if (argc < 3) {
    std::cerr << "Usage: " << argv[0] << " <matrix file> <matrix file>"
              << std::endl;
    return 1;
  }

  try {
    Matrix<data_type> matrix1(argv[1]);
    Matrix<data_type> matrix2(argv[2]);

    Matrix<data_type> answer(matrix1.rows, matrix2.cols);

    auto start = std::chrono::high_resolution_clock::now();
    data_type *device_1, *device_2, *device_answer;

    // auto start_alloc = std::chrono::high_resolution_clock::now();
    // allocate device memory

    hipMalloc(&device_1, matrix1.rows * matrix1.cols * sizeof(data_type));
    hipMalloc(&device_2, matrix2.rows * matrix2.cols * sizeof(data_type));
    hipMalloc(&device_answer, matrix1.rows * matrix2.cols * sizeof(data_type));

    // auto end_alloc = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double, std::micro> duration_alloc =
    //     end_alloc - start_alloc;
    // std::cout << "alloc us: " << duration_alloc.count() << std::endl;

    // copy data to device

    // auto start_copy = std::chrono::high_resolution_clock::now();
    hipMemcpy(device_1, matrix1.begin(),
               matrix1.rows * matrix1.cols * sizeof(data_type),
               hipMemcpyHostToDevice);

    hipMemcpy(device_2, matrix2.begin(),
               matrix2.rows * matrix2.cols * sizeof(data_type),
               hipMemcpyHostToDevice);

    // auto end_copy = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double, std::micro> duration_copy =
    //     end_copy - start_copy;
    // std::cout << "copy us: " << duration_copy.count() << std::endl;

    auto block = 32;
    if (argc > 4) {
      block = std::stoi(argv[4]);
      if (block == 0) {
        block = 32;
      }
    }

    auto grid = (matrix1.cols + block - 1) / block;
    if (argc > 3) {
      grid = std::stoi(argv[3]);
      if (grid == 0) {
        grid = 32;
      }
    }

    dim3 blockSize(block, block);
    dim3 gridSize(grid, grid);

    // start calculation
    // auto start_compute = std::chrono::high_resolution_clock::now();

#ifdef SHARED
    auto name = "shared";
    matrixMulSharedKernel<<<gridSize, blockSize>>>(device_1, device_2,
                                                   device_answer, matrix1.rows,
                                                   matrix1.cols, matrix2.cols);

#elif defined CUBLAS
    auto name = "cublas";

    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);
    data_type alpha = 1.0f;
    data_type beta = 0.0f;

    hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix1.rows, matrix2.cols,
                matrix1.cols, &alpha, device_1, matrix1.cols, device_2,
                matrix2.rows, &beta, device_answer, matrix1.rows);
#else
    auto name = "naive";
    matrixMulKernel<<<gridSize, blockSize>>>(device_1, device_2, device_answer,
                                             matrix1.rows, matrix1.cols,
                                             matrix2.cols);

#endif

    // auto end_compute = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double, std::micro> duration_compute =
    //     end_compute - start_compute;
    // std::cout << "compute us: " << duration_compute.count() << std::endl;

    // copy data back to host
    // auto start_comm = std::chrono::high_resolution_clock::now();
    hipMemcpy(answer.begin(), device_answer,
               matrix1.rows * matrix2.cols * sizeof(int32_t),
               hipMemcpyDeviceToHost);

    // auto end_comm = std::chrono::high_resolution_clock::now();
    // std::chrono::duration<double, std::micro> duration_comm =
    //     end_comm - start_comm;
    // std::cout << "comm us: " << duration_comm.count() << std::endl;

    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::micro> duration = end - start;

    auto size =
        std::to_string(matrix1.rows) + "x" + std::to_string(matrix2.cols);

    std::cout << std::fixed << std::setprecision(0) << name << " " << std::left
              << std::setw(11) << size << " "
              << "grid: " << std::setw(2) << grid << " block: " << std::setw(5)
              << block << " " << duration.count() << std::endl;

    // std::cout << answer << std::endl;

  } catch (const std::exception &e) {
    std::cerr << "Exception: " << e.what() << std::endl;
    return 1;
  }
}
